#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iostream>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <vector>

#include "../include/lie.cuh"
#include "../include/print.cuh"
#include "../include/timer.cuh"

long int get_row_size(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    long i = 0;
    while (f.get(c))
        if (c == '\n')
            ++i;
    f.close();
    return i;
}

enum ColumnT { U64, U32 };

column_type *get_relation_from_file(const char *file_path, int total_rows,
                                    int total_columns, char separator,
                                    ColumnT ct) {
    column_type *data =
        (column_type *)malloc(total_rows * total_columns * sizeof(column_type));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                if (ct == U64) {
                    fscanf(data_file, "%lld%c", &data[(i * total_columns) + j],
                           &separator);
                } else {
                    fscanf(data_file, "%ld%c", &data[(i * total_columns) + j],
                           &separator);
                }
            } else {
                if (ct == U64) {
                    fscanf(data_file, "%lld", &data[(i * total_columns) + j]);
                } else {
                    fscanf(data_file, "%ld", &data[(i * total_columns) + j]);
                }
            }
        }
    }
    return data;
}

__device__ void reorder_path(tuple_type inner, tuple_type outer,
                             tuple_type newt) {
    newt[0] = inner[1];
    newt[1] = outer[1];
};
__device__ tuple_generator_hook reorder_path_device = reorder_path;

int main(int argc, char *argv[]) {
    auto dataset_path = argv[1];
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount,
                           device_id);
    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block,
                           hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "num of sm " << number_of_sm << " num of thread per block "
              << max_threads_per_block << std::endl;
    std::cout << "using " << EMPTY_HASH_ENTRY << " as empty hash entry"
              << std::endl;
    ;
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    std::locale loc("");

    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    time_point_begin = std::chrono::high_resolution_clock::now();
    double spent_time;
    KernelTimer timer;

    // load the raw graph
    tuple_size_t graph_edge_counts = get_row_size(dataset_path);
    std::cout << "Input graph rows: " << graph_edge_counts << std::endl;
    // u64 graph_edge_counts = 2100;
    column_type *raw_graph_data =
        get_relation_from_file(dataset_path, graph_edge_counts, 2, '\t', U32);
    column_type *raw_reverse_graph_data =
        (column_type *)malloc(graph_edge_counts * 2 * sizeof(column_type));
    std::cout << "reversing graph ... " << std::endl;
    for (tuple_size_t i = 0; i < graph_edge_counts; i++) {
        raw_reverse_graph_data[i * 2 + 1] = raw_graph_data[i * 2];
        raw_reverse_graph_data[i * 2] = raw_graph_data[i * 2 + 1];
    }
    std::cout << "finish reverse graph." << std::endl;

    int REPEAT = 1;
    // init the tuples
    time_point_end = std::chrono::high_resolution_clock::now();
    spent_time = std::chrono::duration_cast<std::chrono::duration<double>>(
                     time_point_end - time_point_begin)
                     .count();
    std::cout << "init tuples time: " << spent_time << std::endl;
    column_type *tuple_hashvs;
    hipMalloc((void **)&tuple_hashvs, graph_edge_counts * sizeof(column_type));
    column_type *col_tmp;
    hipMalloc((void **)&col_tmp, graph_edge_counts * sizeof(column_type));

    // load raw data into edge relation
    time_point_begin = std::chrono::high_resolution_clock::now();
    Relation *edge_2__2_1 = new Relation();
    // hipHostMalloc((void **)&edge_2__2_1, sizeof(Relation));
    Relation *path_2__1_2 = new Relation();
    path_2__1_2->index_flag = false;
    // hipHostMalloc((void **)&path_2__1_2, sizeof(Relation));
    std::cout << "edge size " << graph_edge_counts << std::endl;
    load_relation(path_2__1_2, "path_2__1_2", 2, raw_graph_data,
                  graph_edge_counts, 1, 0, grid_size, block_size);
    load_relation(edge_2__2_1, "edge_2__2_1", 2, raw_reverse_graph_data,
                  graph_edge_counts, 1, 0, grid_size, block_size);
    LIE tc_scc(grid_size, block_size);
    tc_scc.max_iteration = 1;
    tc_scc.reload_full_flag = false;
    tc_scc.add_relations(edge_2__2_1, true);
    tc_scc.add_relations(path_2__1_2, false);
    float join_detail[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
    tuple_generator_hook reorder_path_host;
    hipMemcpyFromSymbol(&reorder_path_host, HIP_SYMBOL(reorder_path_device),
                         sizeof(tuple_generator_hook));
    tuple_copy_hook cp_1_host;
    RelationalJoin join_op(edge_2__2_1, FULL, path_2__1_2, DELTA, path_2__1_2,
                           reorder_path_host, nullptr, LEFT, grid_size,
                           block_size, join_detail);
    tc_scc.add_ra(join_op);
    timer.start_timer();
    tc_scc.fixpoint_loop();
    timer.stop_timer();
    std::cout << "Path counts " << path_2__1_2->full->tuple_counts << std::endl;
    // print_tuple_rows(path_2__2_1->full, "full");
    std::cout << "TC time: " << timer.get_spent_time() << std::endl;
    std::cout << "join detail: " << std::endl;
    std::cout << "compute size time:  " << join_detail[0] << std::endl;
    std::cout << "reduce + scan time: " << join_detail[1] << std::endl;
    std::cout << "fetch result time:  " << join_detail[2] << std::endl;
    std::cout << "sort time:          " << join_detail[3] << std::endl;
    std::cout << "build index time:   " << join_detail[5] << std::endl;
    std::cout << "merge time:         " << join_detail[6] << std::endl;
    std::cout << "unique time:        " << join_detail[4] + join_detail[7]
              << std::endl;

    join_op();
    print_memory_usage();
    // deduplicate with full
    time_point_begin = std::chrono::high_resolution_clock::now();
    std::cout << "start deduplicate with full ..." << std::endl;
    tuple_type *dedup_buf;
    hipMalloc((void **)&dedup_buf,
               path_2__1_2->current_full_size * sizeof(tuple_type));
    hipDeviceSynchronize();
    tuple_type *dedup_buf_end = thrust::set_difference(
        thrust::device, path_2__1_2->newt->tuples,
        path_2__1_2->newt->tuples + path_2__1_2->newt->tuple_counts,
        path_2__1_2->tuple_full,
        path_2__1_2->tuple_full + path_2__1_2->current_full_size, dedup_buf,
        tuple_indexed_less(path_2__1_2->full->index_column_size,
                           path_2__1_2->full->arity -
                               path_2__1_2->dependent_column_size));
    hipDeviceSynchronize();
    tuple_size_t tp_counts = dedup_buf_end - dedup_buf;
    time_point_end = std::chrono::high_resolution_clock::now();
    spent_time = std::chrono::duration_cast<std::chrono::duration<double>>(
                     time_point_end - time_point_begin)
                     .count();
    std::cout << "deduplicate with full time: " << spent_time << std::endl;

    // test merge speed
    time_point_begin = std::chrono::high_resolution_clock::now();
    tuple_type *merge_buf;
    std::cout << "start merge test ..." << std::endl;
    timer.start_timer();
    hipMalloc((void **)&merge_buf, path_2__1_2->full->tuple_counts +
                                        tp_counts * sizeof(tuple_type));
    print_memory_usage();
    hipDeviceSynchronize();
    thrust::merge(thrust::device, path_2__1_2->tuple_full,
                  path_2__1_2->tuple_full + path_2__1_2->current_full_size,
                  dedup_buf, dedup_buf_end, merge_buf);
    timer.stop_timer();
    std::cout << "merge int once time: " << timer.get_spent_time() << std::endl;

    std::cout << "start multi merge test ..." << std::endl;
    tuple_size_t merge_step = 5000;
    time_point_begin = std::chrono::high_resolution_clock::now();
    for(tuple_size_t i = 0; i < path_2__1_2->full->tuple_counts; i += merge_step) {
        tuple_size_t merge_size = merge_step;
        if (i + merge_step > path_2__1_2->full->tuple_counts) {
            merge_size = path_2__1_2->full->tuple_counts - i;
        }
        hipDeviceSynchronize();
        thrust::merge(thrust::device, path_2__1_2->tuple_full + i,
                      path_2__1_2->tuple_full + i + merge_size,
                      dedup_buf, dedup_buf_end, merge_buf);
    }
    hipDeviceSynchronize();
    time_point_end = std::chrono::high_resolution_clock::now();
    spent_time = std::chrono::duration_cast<std::chrono::duration<double>>(
                     time_point_end - time_point_begin)
                     .count();
    std::cout << "multi merge time: " << spent_time << std::endl;
    hipFree(merge_buf);
    return 0;
}
