#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iostream>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <vector>

#define EMPTY_HASH_ENTRY ULLONG_MAX

using u64 = unsigned long long;
using u32 = unsigned long;

using column_type = u32;
using tuple_type = column_type *;
using tuple_size_t = u64;
using t_data_internal = u64 *;

typedef void (*tuple_generator_hook)(tuple_type, tuple_type, tuple_type);
typedef void (*tuple_copy_hook)(tuple_type, tuple_type);
typedef bool (*tuple_predicate)(tuple_type);

// struct tuple_generator_hook {
//     __host__ __device__
//     void operator()(tuple_type inner, tuple_type outer, tuple_type newt) {};
// };

/**
 * @brief TODO: remove this use comparator function
 *
 * @param t1
 * @param t2
 * @param l
 * @return true
 * @return false
 */
__host__ __device__ inline bool tuple_eq(tuple_type t1, tuple_type t2,
                                         tuple_size_t l) {
    for (int i = 0; i < l; i++) {
        if (t1[i] != t2[i]) {
            return false;
        }
    }
    return true;
}

struct t_equal {
    u64 arity;

    t_equal(tuple_size_t arity) { this->arity = arity; }

    __host__ __device__ bool operator()(const tuple_type &lhs,
                                        const tuple_type &rhs) {
        for (int i = 0; i < arity; i++) {
            if (lhs[i] != rhs[i]) {
                return false;
            }
        }
        return true;
    }
};

/**
 * @brief fnv1-a hash used in original slog backend
 *
 * @param start_ptr
 * @param prefix_len
 * @return __host__ __device__
 */
__host__ __device__ inline u64 prefix_hash(tuple_type start_ptr,
                                           u64 prefix_len) {
    const u64 base = 14695981039346656037ULL;
    const u64 prime = 1099511628211ULL;

    u64 hash = base;
    for (u64 i = 0; i < prefix_len; ++i) {
        u64 chunk = (u64)start_ptr[i];
        hash ^= chunk & 255ULL;
        hash *= prime;
        for (char j = 0; j < 7; ++j) {
            chunk = chunk >> 8;
            hash ^= chunk & 255ULL;
            hash *= prime;
        }
    }
    return hash;
}

// 32 bit version of fnv1-a
__host__ __device__ inline u32 prefix_hash_32(tuple_type start_ptr,
                                              u64 prefix_len) {
    const u32 base = 2166136261U;
    const u32 prime = 16777619U;

    u32 hash = base;
    for (u64 i = 0; i < prefix_len; ++i) {
        u32 chunk = (u32)start_ptr[i];
        hash ^= chunk & 255U;
        hash *= prime;
        for (char j = 0; j < 3; ++j) {
            chunk = chunk >> 8;
            hash ^= chunk & 255U;
            hash *= prime;
        }
    }
    return hash;
}

// 32bit xxhash version prefix hash
__host__ __device__ inline u32 prefix_hash_xxhash_32(tuple_type start_ptr,
                                                     u64 prefix_len) {
    const u32 prime = 2654435761U;
    u32 hash = 0;
    for (u64 i = 0; i < prefix_len; ++i) {
        u32 chunk = (u32)start_ptr[i];
        hash += chunk * prime;
        hash += (hash << 13);
        hash ^= (hash >> 7);
        hash += (hash << 3);
        hash ^= (hash >> 17);
        hash += (hash << 5);
    }
    return hash;
}

// change to std
struct tuple_indexed_less {

    // u64 *index_columns;
    tuple_size_t index_column_size;
    int arity;

    tuple_indexed_less(tuple_size_t index_column_size, int arity) {
        // this->index_columns = index_columns;
        this->index_column_size = index_column_size;
        this->arity = arity;
    }

    __host__ __device__ bool operator()(const tuple_type &lhs,
                                        const tuple_type &rhs) {
        // fetch the index
        // compare hash first, could be index very different but share the same
        // hash
        if (prefix_hash(lhs, index_column_size) ==
            prefix_hash(rhs, index_column_size)) {
            // same hash
            for (tuple_size_t i = 0; i < arity; i++) {
                if (lhs[i] < rhs[i]) {
                    return true;
                } else if (lhs[i] > rhs[i]) {
                    return false;
                }
            }
            return false;
        } else if (prefix_hash_xxhash_32(lhs, index_column_size) <
                   prefix_hash_xxhash_32(rhs, index_column_size)) {
            return true;
        } else {
            return false;
        }
    }
};

struct tuple_weak_less {

    int arity;

    tuple_weak_less(int arity) { this->arity = arity; }

    __host__ __device__ bool operator()(const tuple_type &lhs,
                                        const tuple_type &rhs) {

        for (u64 i = 0; i < arity; i++) {
            if (lhs[i] < rhs[i]) {
                return true;
            } else if (lhs[i] > rhs[i]) {
                return false;
            }
        }
        return false;
    };
};

long int get_row_size(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    long i = 0;
    while (f.get(c))
        if (c == '\n')
            ++i;
    f.close();
    return i;
}

enum ColumnT { U64, U32 };

column_type *get_relation_from_file(const char *file_path, int total_rows,
                                    int total_columns, char separator,
                                    ColumnT ct) {
    column_type *data =
        (column_type *)malloc(total_rows * total_columns * sizeof(column_type));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                if (ct == U64) {
                    fscanf(data_file, "%lld%c", &data[(i * total_columns) + j],
                           &separator);
                } else {
                    fscanf(data_file, "%ld%c", &data[(i * total_columns) + j],
                           &separator);
                }
            } else {
                if (ct == U64) {
                    fscanf(data_file, "%lld", &data[(i * total_columns) + j]);
                } else {
                    fscanf(data_file, "%ld", &data[(i * total_columns) + j]);
                }
            }
        }
    }
    return data;
}

// print tuples
void print_tuple_list(tuple_type *tuples, tuple_size_t rows,
                      tuple_size_t arity) {
    tuple_type *tuples_host;
    hipHostMalloc((void **)&tuples_host, rows * sizeof(tuple_type));
    hipMemcpy(tuples_host, tuples, rows * sizeof(tuple_type),
               hipMemcpyDeviceToHost);
    if (rows > 100) {
        rows = 100;
    }
    for (tuple_size_t i = 0; i < rows; i++) {
        tuple_type cur_tuple = tuples_host[i];

        tuple_type cur_tuple_host;
        hipHostMalloc((void **)&cur_tuple_host, arity * sizeof(column_type));
        hipMemcpy(cur_tuple_host, cur_tuple, arity * sizeof(column_type),
                   hipMemcpyDeviceToHost);
        for (tuple_size_t j = 0; j < arity; j++) {
            std::cout << cur_tuple_host[j] << " ";
        }
        std::cout << std::endl;
    }
}

// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>

// Number of bits per pass
const int BITS_PER_PASS = 4;

// Number of bins per pass
const int BINS_PER_PASS = 1 << BITS_PER_PASS;

// Number of threads per block
const int THREADS_PER_BLOCK = 256;

// Radix sort kernel
__global__ void radix_sort_kernel(u32 *data, int *temp, int *histogram,
                                  int num_elements, int pass) {
    // Compute the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute the local thread ID within the warp
    int lane = threadIdx.x & 31;

    // Compute the histogram index for this thread
    int index = (data[tid] >> (pass * BITS_PER_PASS)) & (BINS_PER_PASS - 1);

    // Compute the starting index for this bin in the temp array
    int start = histogram[index * blockDim.x + lane];

    // Compute the ending index for this bin in the temp array
    int end = start + histogram[index * blockDim.x + blockDim.x - 1];

    // Copy the element to the temp array
    temp[start + lane] = data[tid];

    // Increment the histogram count for this bin
    atomicAdd(&histogram[index * blockDim.x + lane], 1);

    // Wait for all threads to finish updating the histogram
    __syncthreads();

    // Compute the starting index for this thread's bin in the temp array
    start = histogram[index * blockDim.x + lane];

    // Copy the element to the temp array
    temp[start + lane] = data[tid];

    // Wait for all threads to finish copying to the temp array
    __syncthreads();

    // Update the data array with the sorted elements
    data[tid] = temp[tid];
}

// Radix sort function
void radix_sort(column_type *data, int arity, int num_elements) {
    // Allocate memory for the temp array and histogram
    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    int *temp, *histogram;
    hipMalloc(&temp, num_elements * sizeof(int));
    hipMalloc(&histogram, BINS_PER_PASS * THREADS_PER_BLOCK * sizeof(int));

    // Initialize the histogram to zero
    hipMemset(histogram, 0, BINS_PER_PASS * THREADS_PER_BLOCK * sizeof(int));
    column_type pass_cnt = sizeof(column_type) * 8 * arity / BITS_PER_PASS;

    // Perform the radix sort passes
    for (column_type pass = 0; pass < pass_cnt; pass++) {
        // Launch the radix sort kernel
        radix_sort_kernel<<<(num_elements + THREADS_PER_BLOCK - 1) /
                                THREADS_PER_BLOCK,
                            THREADS_PER_BLOCK>>>(data+arity, temp, histogram,
                                                 num_elements, pass);

        // Clear the histogram for the next pass
        hipMemset(histogram, 0,
                   BINS_PER_PASS * THREADS_PER_BLOCK * sizeof(int));
    }

    // Free the memory
    hipFree(temp);
    hipFree(histogram);
}

__global__ void init_tuples_unsorted(tuple_type *tuples, column_type *raw_data,
                                     int arity, tuple_size_t rows) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= rows)
        return;

    int stride = blockDim.x * gridDim.x;
    for (tuple_size_t i = index; i < rows; i += stride) {
        tuples[i] = raw_data + i * arity;
    }
}

// cuda kernel compute hash for tuples using fnv1-a
__global__ void compute_hash(tuple_type *tuples, tuple_size_t rows,
                             tuple_size_t index_column_size,
                             column_type *hashes) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= rows)
        return;

    int stride = blockDim.x * gridDim.x;
    for (tuple_size_t i = index; i < rows; i += stride) {
        hashes[i] =
            (column_type)prefix_hash_xxhash_32(tuples[i], index_column_size);
    }
}

// cuda kernel extract the k th column from tuples
__global__ void extract_column(tuple_type *tuples, tuple_size_t rows,
                               tuple_size_t k, column_type *column) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= rows)
        return;

    int stride = blockDim.x * gridDim.x;
    for (tuple_size_t i = index; i < rows; i += stride) {
        column[i] = tuples[i][k];
    }
}

int main(int argc, char *argv[]) {
    auto dataset_path = argv[1];
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount,
                           device_id);
    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "num of sm " << number_of_sm << " num of thread per block " << max_threads_per_block << std::endl;
    std::cout << "using " << EMPTY_HASH_ENTRY << " as empty hash entry"
              << std::endl;;
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    std::locale loc("");

    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    time_point_begin = std::chrono::high_resolution_clock::now();
    double spent_time;

    // load the raw graph
    tuple_size_t graph_edge_counts = get_row_size(dataset_path);
    std::cout << "Input graph rows: " << graph_edge_counts << std::endl;
    // u64 graph_edge_counts = 2100;
    column_type *raw_graph_data =
        get_relation_from_file(dataset_path, graph_edge_counts, 2, '\t', U32);
    column_type *raw_reverse_graph_data =
        (column_type *)malloc(graph_edge_counts * 2 * sizeof(column_type));
    std::cout << "reversing graph ... " << std::endl;
    for (tuple_size_t i = 0; i < graph_edge_counts; i++) {
        raw_reverse_graph_data[i * 2 + 1] = raw_graph_data[i * 2];
        raw_reverse_graph_data[i * 2] = raw_graph_data[i * 2 + 1];
    }
    std::cout << "finish reverse graph." << std::endl;

    // copy the graph to device
    column_type *d_graph_data;
    hipMalloc((void **)&d_graph_data,
               graph_edge_counts * relation_columns * sizeof(column_type));
    hipMemcpy(d_graph_data, raw_graph_data,
               graph_edge_counts * relation_columns * sizeof(column_type),
               hipMemcpyHostToDevice);

    int REPEAT = 1;
    // init the tuples
    tuple_type *tuples;
    hipMalloc(&tuples, graph_edge_counts * sizeof(tuple_type));
    time_point_begin = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < REPEAT; i++) {
        init_tuples_unsorted<<<grid_size, block_size>>>(
            tuples, d_graph_data, relation_columns, graph_edge_counts);
    }
    hipDeviceSynchronize();
    time_point_end = std::chrono::high_resolution_clock::now();
    spent_time = std::chrono::duration_cast<std::chrono::duration<double>>(
                     time_point_end - time_point_begin)
                     .count();
    std::cout << "init tuples time: " << spent_time << std::endl;
    column_type *tuple_hashvs;
    hipMalloc((void **)&tuple_hashvs, graph_edge_counts * sizeof(column_type));
    column_type *col_tmp;
    hipMalloc((void **)&col_tmp, graph_edge_counts * sizeof(column_type));

    time_point_end = std::chrono::high_resolution_clock::now();
    // compute hash for tuples
    for (int i = 0; i < REPEAT; i++) {
        compute_hash<<<grid_size, block_size>>>(tuples, graph_edge_counts, 1,
                                                tuple_hashvs);
        hipDeviceSynchronize();
    }
    time_point_end = std::chrono::high_resolution_clock::now();
    spent_time = std::chrono::duration_cast<std::chrono::duration<double>>(
                     time_point_end - time_point_begin)
                     .count();
    std::cout << "compute hash time: " << spent_time << std::endl;

    // sort the tuples using thrust
    double sort_hash_time = 0;
    for (int i = 0; i < REPEAT; i++) {
        time_point_begin = std::chrono::high_resolution_clock::now();

        extract_column<<<grid_size, block_size>>>(tuples, graph_edge_counts, 1,
                                                  col_tmp);
        hipDeviceSynchronize();
        thrust::stable_sort_by_key(thrust::device, col_tmp,
                                   col_tmp + graph_edge_counts, tuples);
        hipDeviceSynchronize();
        extract_column<<<grid_size, block_size>>>(tuples, graph_edge_counts, 0,
                                                  col_tmp);
        hipDeviceSynchronize();
        thrust::stable_sort_by_key(thrust::device, col_tmp,
                                   col_tmp + graph_edge_counts, tuples);
        compute_hash<<<grid_size, block_size>>>(tuples, graph_edge_counts, 1,
                                                tuple_hashvs);
        hipDeviceSynchronize();
        thrust::stable_sort_by_key(thrust::device, tuple_hashvs,
                                   tuple_hashvs + graph_edge_counts, tuples);
        hipDeviceSynchronize();
        time_point_end = std::chrono::high_resolution_clock::now();
        sort_hash_time +=
            std::chrono::duration_cast<std::chrono::duration<double>>(
                time_point_end - time_point_begin)
                .count();
        print_tuple_list(tuples, graph_edge_counts, 2);
        // recover prepare for next sort
        init_tuples_unsorted<<<grid_size, block_size>>>(
            tuples, d_graph_data, relation_columns, graph_edge_counts);
    }
    std::cout << "sort hash time: " << sort_hash_time << std::endl;

    // sort the tuples using thrust with tuple_indexed_less
    double sort_comp_time = 0;
    for (int i = 0; i < REPEAT; i++) {
        time_point_begin = std::chrono::high_resolution_clock::now();
        thrust::sort(thrust::device, tuples, tuples + graph_edge_counts,
                     tuple_indexed_less(1, 2));
        hipDeviceSynchronize();
        time_point_end = std::chrono::high_resolution_clock::now();
        sort_comp_time +=
            std::chrono::duration_cast<std::chrono::duration<double>>(
                time_point_end - time_point_begin)
                .count();
        print_tuple_list(tuples, graph_edge_counts, 2);
        init_tuples_unsorted<<<grid_size, block_size>>>(
            tuples, d_graph_data, relation_columns, graph_edge_counts);
    }
    std::cout << "sort using tuple_indexed_less time: " << sort_comp_time
              << std::endl;

    return 0;
}
