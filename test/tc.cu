#include <chrono>
#include <fstream>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <vector>

#include "../include/exception.cuh"
#include "../include/lie.cuh"
#include "../include/timer.cuh"


//////////////////////////////////////////////////////

long int get_row_size(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    long i = 0;
    while (f.get(c))
        if (c == '\n')
            ++i;
    f.close();
    return i;
}

column_type *get_relation_from_file(const char *file_path, int total_rows,
                                    int total_columns, char separator) {
    column_type *data =
        (column_type *)malloc(total_rows * total_columns * sizeof(column_type));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%lld%c", &data[(i * total_columns) + j],
                       &separator);
            } else {
                fscanf(data_file, "%lld", &data[(i * total_columns) + j]);
            }
        }
    }
    return data;
}

//////////////////////////////////////////////////////////////////

__device__ void reorder_path(tuple_type inner, tuple_type outer,
                                    tuple_type newt) {
    newt[0] = inner[1];
    newt[1] = outer[1];
};
__device__ tuple_generator_hook reorder_path_device = reorder_path;

void graph_bench(const char *dataset_path, int block_size, int grid_size) {
    KernelTimer timer;
    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    time_point_begin = std::chrono::high_resolution_clock::now();
    double spent_time;

    // load the raw graph
    u64 graph_edge_counts = get_row_size(dataset_path);
    std::cout << "Input graph rows: " << graph_edge_counts << std::endl;
    // u64 graph_edge_counts = 2100;
    column_type *raw_graph_data =
        get_relation_from_file(dataset_path, graph_edge_counts, 2, '\t');
    column_type *raw_reverse_graph_data =
        (column_type *)malloc(graph_edge_counts * 2 * sizeof(column_type));

    std::cout << "reversing graph ... " << std::endl;
    for (u64 i = 0; i < graph_edge_counts; i++) {
        raw_reverse_graph_data[i * 2 + 1] = raw_graph_data[i * 2];
        raw_reverse_graph_data[i * 2] = raw_graph_data[i * 2 + 1];
    }
    std::cout << "finish reverse graph." << std::endl;

    timer.start_timer();
    Relation *edge_2__2_1;
    hipHostMalloc((void **)&edge_2__2_1, sizeof(Relation));
    Relation *path_2__1_2;
    hipHostMalloc((void **)&path_2__1_2, sizeof(Relation));
    std::cout << "edge size " << graph_edge_counts << std::endl;
    load_relation(path_2__1_2, "path_2__1_2", 2, raw_graph_data,
                  graph_edge_counts, 1, grid_size, block_size);
    load_relation(edge_2__2_1, "edge_2__2_1", 2, raw_reverse_graph_data,
                  graph_edge_counts, 1, grid_size, block_size);
    timer.stop_timer();
    // double kernel_spent_time = timer.get_spent_time();
    std::cout << "Build hash table time: " << timer.get_spent_time()
              << std::endl;

    timer.start_timer();
    LIE tc_scc(grid_size, block_size);
    tc_scc.add_relations(edge_2__2_1, true);
    tc_scc.add_relations(path_2__1_2, false);
    float join_time[3];
    tuple_generator_hook reorder_path_host;
    hipMemcpyFromSymbol(&reorder_path_host, HIP_SYMBOL(reorder_path_device), sizeof(tuple_generator_hook));
    tc_scc.add_ra(RelationalJoin(edge_2__2_1, FULL, path_2__1_2, DELTA,
                                 path_2__1_2, reorder_path_host, LEFT, grid_size,
                                 block_size, join_time));
    tc_scc.fixpoint_loop();

    timer.stop_timer();
    std::cout << "TC time: " << timer.get_spent_time() << std::endl;
}

int main(int argc, char *argv[]) {
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount,
                           device_id);
    std::cout << "num of sm " << number_of_sm << std::endl;
    std::cout << "using " << EMPTY_HASH_ENTRY << " as empty hash entry"
              << std::endl;
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    std::locale loc("");

    graph_bench(argv[1], block_size, grid_size);
    return 0;
}