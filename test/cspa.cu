#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <vector>

#include "../include/exception.cuh"
#include "../include/lie.cuh"
#include "../include/print.cuh"
#include "../include/timer.cuh"

//////////////////////////////////////////////////////

long int get_row_size(const char *data_path) {
    std::ifstream f;
    f.open(data_path);
    char c;
    long i = 0;
    while (f.get(c))
        if (c == '\n')
            ++i;
    f.close();
    return i;
}

enum ColumnT { U64, U32 };

column_type *get_relation_from_file(const char *file_path, int total_rows,
                                    int total_columns, char separator,
                                    ColumnT ct) {
    column_type *data =
        (column_type *)malloc(total_rows * total_columns * sizeof(column_type));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                if (ct == U64) {
                    fscanf(data_file, "%lld%c", &data[(i * total_columns) + j],
                           &separator);
                } else {
                    fscanf(data_file, "%ld%c", &data[(i * total_columns) + j],
                           &separator);
                }
            } else {
                if (ct == U64) {
                    fscanf(data_file, "%lld", &data[(i * total_columns) + j]);
                } else {
                    fscanf(data_file, "%ld", &data[(i * total_columns) + j]);
                }
            }
        }
    }
    return data;
}

//////////////////////////////////////////////////////////////////

__device__ void cp_2_1__1(tuple_type input, tuple_type outpt) {
    outpt[0] = input[0];
    outpt[1] = input[0];
};
__device__ tuple_copy_hook cp_2_1__1_device = cp_2_1__1;
__device__ void cp_2_1__2(tuple_type input, tuple_type outpt) {
    outpt[0] = input[1];
    outpt[1] = input[1];
};
__device__ tuple_copy_hook cp_2_1__2_device = cp_2_1__2;

__device__ void cp_2_1__1_2(tuple_type input, tuple_type outpt) {
    outpt[0] = input[1];
    outpt[1] = input[0];
};
__device__ tuple_copy_hook cp_2_1__1_2_device = cp_2_1__1_2;
__device__ void cp_2_1__2_1(tuple_type input, tuple_type outpt) {
    outpt[0] = input[0];
    outpt[1] = input[1];
};
__device__ tuple_copy_hook cp_2_1__2_1_device = cp_2_1__2_1;

__device__ void join_10_11(tuple_type inner, tuple_type outer,
                           tuple_type output) {
    output[1] = inner[1];
    output[0] = outer[1];
}
__device__ tuple_generator_hook join_10_11_device = join_10_11;

__device__ void join_01_11(tuple_type inner, tuple_type outer,
                           tuple_type output) {
    output[0] = inner[1];
    output[1] = outer[1];
}
__device__ tuple_generator_hook join_01_11_device = join_01_11;

////////////////////////////////////////////////////////////////

void analysis_bench(const char *dataset_path, int block_size, int grid_size) {
    KernelTimer timer;
    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    
    double spent_time;

    // load the input relation
    std::stringstream assign_fact_ss;
    assign_fact_ss << dataset_path << "/assign.facts";
    std::stringstream dereference_fact_ss;
    dereference_fact_ss << dataset_path << "/dereference.facts";
    // std::cout << assign_fact_ss.str() << std::endl;
    tuple_size_t assign_counts = get_row_size(assign_fact_ss.str().c_str());
    std::cout << "Input assign rows: " << assign_counts << std::endl;
    column_type *raw_assign_data = get_relation_from_file(
        assign_fact_ss.str().c_str(), assign_counts, 2, '\t', U32);
    std::cout << "reversing assign ... " << std::endl;
    column_type *raw_reverse_assign_data =
        (column_type *)malloc(assign_counts * 2 * sizeof(column_type));
    for (tuple_size_t i = 0; i < assign_counts; i++) {
        raw_reverse_assign_data[i * 2 + 1] = raw_assign_data[i * 2];
        raw_reverse_assign_data[i * 2] = raw_assign_data[i * 2 + 1];
    }

    tuple_size_t dereference_counts =
        get_row_size(dereference_fact_ss.str().c_str());
    std::cout << "Input dereference rows: " << dereference_counts << std::endl;
    column_type *raw_dereference_data = get_relation_from_file(
        dereference_fact_ss.str().c_str(), dereference_counts, 2, '\t', U32);
    std::cout << "reversing dereference ... " << std::endl;
    column_type *raw_reverse_dereference_data =
        (column_type *)malloc(dereference_counts * 2 * sizeof(column_type));
    for (tuple_size_t i = 0; i < dereference_counts; i++) {
        raw_reverse_dereference_data[i * 2 + 1] = raw_dereference_data[i * 2];
        raw_reverse_dereference_data[i * 2] = raw_dereference_data[i * 2 + 1];
    }

    timer.start_timer();
    
    Relation *assign_2__2_1 = new Relation();
    load_relation(assign_2__2_1, "assign_2__2_1", 2, raw_reverse_assign_data,
                  assign_counts, 1, 0, grid_size, block_size);

    Relation *dereference_2__1_2 = new Relation();
    load_relation(dereference_2__1_2, "dereference_2__1_2", 2,
                  raw_dereference_data, dereference_counts, 1, 0, grid_size,
                  block_size);
    Relation *dereference_2__2_1 = new Relation();
    load_relation(dereference_2__2_1, "dereference_2__2_1", 2,
                  raw_reverse_dereference_data, dereference_counts, 1, 0,
                  grid_size, block_size);
    timer.stop_timer();
    std::cout << "Build hash table time: " << timer.get_spent_time()
              << std::endl;

    // scc init
    Relation *value_flow_2__1_2 = new Relation();
    load_relation(value_flow_2__1_2, "value_flow_2__1_2", 2, nullptr, 0, 1, 0,
                  grid_size, block_size);
    Relation *value_flow_2__2_1 = new Relation();
    load_relation(value_flow_2__2_1, "value_flow_2__2_1", 2, nullptr, 0, 1, 0,
                  grid_size, block_size);

    Relation *memory_alias_2__1_2 = new Relation();
    load_relation(memory_alias_2__1_2, "memory_alias_2__1_2", 2, nullptr, 0, 1,
                  0, grid_size, block_size);
    Relation *memory_alias_2__2_1 = new Relation();
    load_relation(memory_alias_2__2_1, "memory_alias_2__2_1", 2, nullptr, 0, 1,
                  0, grid_size, block_size);

    timer.start_timer();
    time_point_begin = std::chrono::high_resolution_clock::now();
    LIE init_scc(grid_size, block_size);
    init_scc.add_relations(value_flow_2__1_2, false);
    init_scc.add_relations(value_flow_2__2_1, false);
    init_scc.add_relations(memory_alias_2__1_2, false);
    init_scc.add_relations(memory_alias_2__2_1, false);
    init_scc.add_relations(assign_2__2_1, true);
    tuple_copy_hook cp_2_1__1_host;
    checkCuda(hipMemcpyFromSymbol(&cp_2_1__1_host, HIP_SYMBOL(cp_2_1__1_device),
                         sizeof(tuple_copy_hook)));
    tuple_copy_hook cp_2_1__2_host;
    checkCuda(hipMemcpyFromSymbol(&cp_2_1__2_host, HIP_SYMBOL(cp_2_1__2_device),
                         sizeof(tuple_copy_hook)));
    tuple_copy_hook cp_2_1__1_2_host;
    checkCuda(hipMemcpyFromSymbol(&cp_2_1__1_2_host, HIP_SYMBOL(cp_2_1__1_2_device),
                         sizeof(tuple_copy_hook)));
    tuple_copy_hook cp_2_1__2_1_host;
    checkCuda(hipMemcpyFromSymbol(&cp_2_1__1_host, HIP_SYMBOL(cp_2_1__1_device),
                         sizeof(tuple_copy_hook)));
    init_scc.add_ra(RelationalCopy(assign_2__2_1, FULL, value_flow_2__1_2,
                                   cp_2_1__1_host, nullptr, grid_size,
                                   block_size));
    init_scc.add_ra(RelationalCopy(assign_2__2_1, FULL, value_flow_2__1_2,
                                   cp_2_1__2_host, nullptr, grid_size,
                                   block_size));
    init_scc.add_ra(RelationalCopy(assign_2__2_1, FULL, value_flow_2__1_2,
                                   cp_2_1__1_2_host, nullptr, grid_size,
                                   block_size));

    init_scc.add_ra(RelationalCopy(assign_2__2_1, FULL, memory_alias_2__1_2,
                                   cp_2_1__1_host, nullptr, grid_size,
                                   block_size));
    init_scc.add_ra(RelationalCopy(assign_2__2_1, FULL, memory_alias_2__1_2,
                                   cp_2_1__2_host, nullptr, grid_size,
                                   block_size));

    init_scc.add_ra(RelationalCopy(value_flow_2__1_2, DELTA, value_flow_2__2_1,
                                   cp_2_1__1_2_host, nullptr, grid_size,
                                   block_size));
    init_scc.add_ra(RelationalCopy(memory_alias_2__1_2, DELTA, memory_alias_2__2_1,
                                   cp_2_1__1_2_host, nullptr, grid_size,
                                   block_size));
    init_scc.fixpoint_loop();

    timer.stop_timer();
    time_point_end = std::chrono::high_resolution_clock::now();
    std::cout << "init scc time: " << timer.get_spent_time() << std::endl;
    std::cout << "init scc time (chono): "
              << std::chrono::duration_cast<std::chrono::milliseconds>(
                     time_point_end - time_point_begin)
                     .count()
              << std::endl;

    // scc analysis
    Relation *value_flow_forward_2__1_2 = new Relation();
    load_relation(value_flow_forward_2__1_2, "value_flow_forward_2__1_2", 2,
                  nullptr, 0, 1, 0, grid_size, block_size);

    Relation *value_flow_forward_2__2_1 = new Relation();
    load_relation(value_flow_forward_2__2_1, "value_flow_forward_2__2_1", 2,
                  nullptr, 0, 1, 0, grid_size, block_size);

    Relation *value_alias_2__1_2 = new Relation();
    value_alias_2__1_2->index_flag = false;
    load_relation(value_alias_2__1_2, "value_alias_2__1_2", 2, nullptr, 0, 1, 0,
                  grid_size, block_size);

    Relation *tmp_rel_def = new Relation();
    tmp_rel_def->index_flag = false;
    load_relation(tmp_rel_def, "tmp_rel_def", 2, nullptr, 0, 1, 0, grid_size,
                  block_size);
    Relation *tmp_rel_ma1 = new Relation();
    tmp_rel_ma1->index_flag = false;
    load_relation(tmp_rel_ma1, "tmp_rel_ma1", 2, nullptr, 0, 1, 0, grid_size,
                  block_size, true);
     Relation *tmp_rel_ma2 = new Relation();
    tmp_rel_ma2->index_flag = false;
    load_relation(tmp_rel_ma2, "tmp_rel_ma2", 2, nullptr, 0, 1, 0, grid_size,
                  block_size, true);

    LIE analysis_scc(grid_size, block_size);

    analysis_scc.add_relations(assign_2__2_1, true);
    analysis_scc.add_relations(dereference_2__1_2, true);
    analysis_scc.add_relations(dereference_2__2_1, true);

    analysis_scc.add_relations(value_flow_2__1_2, false);
    analysis_scc.add_relations(value_flow_2__2_1, false);
    analysis_scc.add_relations(memory_alias_2__1_2, false);
    analysis_scc.add_relations(memory_alias_2__2_1, false);
    analysis_scc.add_relations(value_alias_2__1_2, false);

    // join order matters for temp!
    analysis_scc.add_tmp_relation(tmp_rel_def);
    analysis_scc.add_tmp_relation(tmp_rel_ma1);
    analysis_scc.add_tmp_relation(tmp_rel_ma2);

    float join_detail[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    // join_vf_vfvf: ValueFlow(x, y) :- ValueFlow(x, z), ValueFlow(z, y).
    tuple_generator_hook join_10_11_host;
    checkCuda(hipMemcpyFromSymbol(&join_10_11_host, HIP_SYMBOL(join_10_11_device),
                         sizeof(tuple_generator_hook)));
    tuple_generator_hook join_01_11_host;
    checkCuda(hipMemcpyFromSymbol(&join_01_11_host, HIP_SYMBOL(join_01_11_device),
                         sizeof(tuple_generator_hook)));
    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, value_flow_2__2_1, DELTA,
                       value_flow_2__1_2, join_10_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));
    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__2_1, FULL, value_flow_2__1_2, DELTA,
                       value_flow_2__1_2, join_01_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    // join_va_vf_vf: ValueAlias(x, y) :- ValueFlow(z, x), ValueFlow(z, y).
    // v1
    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, value_flow_2__1_2, DELTA,
                       value_alias_2__1_2, join_01_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));
    // v2
    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, value_flow_2__1_2, DELTA,
                       value_alias_2__1_2, join_10_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    // join_vf_am: ValueFlow(x, y) :- Assign(x, z), MemoryAlias(z, y).
    analysis_scc.add_ra(
        RelationalJoin(assign_2__2_1, FULL, memory_alias_2__1_2, DELTA,
                       value_flow_2__1_2, join_01_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    // tmp_rel_def(z, x) :- Dereference(y, x), ValueAlias(y, z)
    analysis_scc.add_ra(
        RelationalJoin(dereference_2__1_2, FULL, value_alias_2__1_2, DELTA,
                       tmp_rel_def, join_10_11_host, nullptr, LEFT, grid_size,
                       block_size, join_detail));

    // WARNING: tmp relation can only in outer because it doesn't include
    // index!
    // join_ma_d_tmp: MemoryAlias(x, w) :- Dereference(z, w) , tmp_rel_def(z,x)
    analysis_scc.add_ra(
        RelationalJoin(dereference_2__1_2, FULL, tmp_rel_def, NEWT,
                       memory_alias_2__1_2, join_10_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    // ValueAlias(x,y) :- 
    //    ValueFlow(z,x),
    //    MemoryAlias(z,w),
    //    ValueFlow(w,y).
    // ValueFlow DELTA 1, 2 <> MemoryAlias FULL 1, 2 <> ValueFlow FULL 2, 1
    // ValueFlow FULL 1, 2 <> MemoryAlias DELTA 1, 2 <> ValueFlow FULL 2, 1
    // ValueFlow FULL 1, 2 <> MemoryAlias FULL 1, 2 <> ValueFlow DELTA 2, 1 
    // join_tmp_vf_ma : tmp_rel_ma(w, x) :- ValueFlow(z, x), MemoryAlias(z, w).
    // join_va_tmp_vf : ValueAlias(x, y) :- tmp_rel_ma(w, x), ValueFlow(w,y).
    // v1
    analysis_scc.add_ra(
        RelationalJoin(memory_alias_2__1_2, FULL , value_flow_2__1_2, DELTA,
                       tmp_rel_ma1, join_01_11_host, nullptr, LEFT, grid_size,
                       block_size, join_detail));
    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, memory_alias_2__1_2, DELTA,
                       tmp_rel_ma1, join_10_11_host, nullptr, LEFT, grid_size,
                       block_size, join_detail));

    analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, tmp_rel_ma1, NEWT,
                       value_alias_2__1_2, join_10_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    analysis_scc.add_ra(
        RelationalJoin(memory_alias_2__2_1, FULL , value_flow_2__1_2, DELTA,
                       tmp_rel_ma2, join_01_11_host, nullptr, LEFT, grid_size,
                       block_size, join_detail));
     analysis_scc.add_ra(
        RelationalJoin(value_flow_2__1_2, FULL, tmp_rel_ma2, NEWT,
                       value_alias_2__1_2, join_01_11_host, nullptr, LEFT,
                       grid_size, block_size, join_detail));

    analysis_scc.add_ra(RelationalACopy(value_flow_2__1_2, value_flow_2__2_1,
                                        cp_2_1__1_2_host, nullptr, grid_size,
                                        block_size));
    analysis_scc.add_ra(RelationalACopy(memory_alias_2__1_2, memory_alias_2__2_1,
                                        cp_2_1__1_2_host, nullptr, grid_size,
                                        block_size));
    time_point_begin = std::chrono::high_resolution_clock::now();
    timer.start_timer();
    analysis_scc.fixpoint_loop();
    // print_tuple_rows(value_flow_2__1_2->full, "value_flow_2__1_2");
    timer.stop_timer();
    time_point_end = std::chrono::high_resolution_clock::now();
    std::cout << "analysis scc time: " << timer.get_spent_time() << std::endl;
    std::cout << "analysis scc time (chono): "
              << std::chrono::duration_cast<std::chrono::milliseconds>(
                     time_point_end - time_point_begin)
                     .count()
              << std::endl;
    std::cout << "join detail: " << std::endl;
    std::cout << "compute size time:  " <<  join_detail[0] <<  std::endl;
    std::cout << "reduce + scan time: " <<  join_detail[1] <<  std::endl;
    std::cout << "fetch result time:  " <<  join_detail[2] <<  std::endl;
    std::cout << "sort time:          " <<  join_detail[3] <<  std::endl;
    std::cout << "build index time:   " <<  join_detail[5] <<  std::endl;
    std::cout << "merge time:         " <<  join_detail[6] <<  std::endl;
    std::cout << "unique time:        " << join_detail[4] + join_detail[7] <<  std::endl;
}

int main(int argc, char *argv[]) {
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount,
                           device_id);
    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "num of sm " << number_of_sm << " num of thread per block " << max_threads_per_block << std::endl;
    std::cout << "using " << EMPTY_HASH_ENTRY << " as empty hash entry"
              << std::endl;
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    std::locale loc("");
    analysis_bench(argv[1], block_size, grid_size);
    return 0;
}
