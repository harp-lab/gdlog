#include "../include/exception.cuh"
#include "../include/print.cuh"
#include <iostream>
#include <rmm/device_vector.hpp>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

void print_hashes(GHashRelContainer *target, const char *rel_name) {
    thrust::host_vector<MEntity> host_map = target->index_map;
    std::cout << "Relation hash >>> " << rel_name << std::endl;
    for (tuple_size_t i = 0; i < target->index_map_size; i++) {
        std::cout << host_map[i].key << "    " << host_map[i].value
                  << std::endl;
    }
    std::cout << "end <<<" << std::endl;
}

void print_tuple_rows(GHashRelContainer *target, const char *rel_name,
                      bool sort_flag) {
    // sort first
    rmm::device_vector<tuple_type> natural_ordered = target->tuples;
    if (sort_flag) {
        thrust::sort(thrust::device, natural_ordered.begin(),
                     natural_ordered.begin() + target->tuple_counts,
                     tuple_weak_less(target->arity));
    }
    // thrust::host_vector<tuple_type> tuples_host = natural_ordered;
    std::cout << "Relation tuples >>> " << rel_name << std::endl;
    std::cout << "Total tuples counts:  " << target->tuple_counts << std::endl;
    u32 pt_size = target->tuple_counts;
    // if (target->tuple_counts > 3000) {
    //     pt_size = 100;
    // }
    for (tuple_size_t i = 0; i < pt_size; i++) {
        tuple_type cur_tuple = natural_ordered[i];
        if (cur_tuple == nullptr) {
            std::cout << "null tuple" << std::endl;
            continue;
        }

        tuple_type cur_tuple_host;
        hipHostMalloc((void **)&cur_tuple_host,
                       target->arity * sizeof(column_type));
        hipMemcpy(cur_tuple_host, cur_tuple,
                   target->arity * sizeof(column_type), hipMemcpyDeviceToHost);
        // if (cur_tuple_host[0] != 1966) {
        //     continue;
        // }
        for (int j = 0; j < target->arity; j++) {
            std::cout << cur_tuple_host[j] << "\t";
        }
        std::cout << std::endl;
        hipHostFree(cur_tuple_host);
    }
    // if (target->tuple_counts > 3000) {
    //     std::cout << "........." << std::endl;
    // }
    std::cout << "end <<<" << std::endl;
}

void print_tuple_raw_data(GHashRelContainer *target, const char *rel_name) {
    std::cout << "Relation raw tuples >>> " << rel_name << std::endl;
    std::cout << "Total raw tuples counts:  " << target->data_raw_row_size
              << std::endl;
    column_type *cur_tuple_host;
    hipHostMalloc((void **)&cur_tuple_host,
                   target->arity * sizeof(column_type));
    for (tuple_size_t i = 0; i < target->data_raw_row_size; i++) {
        hipMemcpy(cur_tuple_host,
                   target->data_raw.data().get() + i * target->arity,
                   target->arity * sizeof(column_type), hipMemcpyDeviceToHost);
        for (int j = 0; j < target->arity; j++) {
            std::cout << cur_tuple_host[j] << "    ";
        }
        std::cout << std::endl;
    }
    hipHostFree(cur_tuple_host);
}

void print_memory_usage() {
    int num_gpus;
    size_t free, total;
    // hipGetDeviceCount( &num_gpus );
    // for ( int gpu_id = 0; gpu_id < num_gpus; gpu_id++ ) {
    // hipSetDevice( gpu_id );
    // int id = 0;
    // hipGetDevice( &id );
    hipMemGetInfo(&free, &total);
    std::cout << "GPU " << 0 << " memory: free=" << free << ", total=" << total
              << std::endl;
    // }
}

tuple_size_t get_free_memory() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return free;
}

tuple_size_t get_total_memory() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return total;
}

// void print_tuple_list(tuple_type* tuples, tuple_size_t rows, tuple_size_t
// arity) {
//     tuple_type* tuples_host;
//     hipHostMalloc((void**) &tuples_host, rows * sizeof(tuple_type));
//     hipMemcpy(tuples_host, tuples, rows * sizeof(tuple_type),
//                hipMemcpyDeviceToHost);
//     if (rows > 100) {
//         rows = 100;
//     }
//     for (tuple_size_t i = 0; i < rows; i++) {
//         tuple_type cur_tuple = tuples_host[i];

//         tuple_type cur_tuple_host;
//         hipHostMalloc((void**) &cur_tuple_host, arity *
//         sizeof(column_type)); hipMemcpy(cur_tuple_host, cur_tuple, arity *
//         sizeof(column_type),
//                    hipMemcpyDeviceToHost);
//         for (tuple_size_t j = 0; j < arity; j++) {
//             std::cout << cur_tuple_host[j] << " ";
//         }
//         std::cout << std::endl;
//     }
// }
