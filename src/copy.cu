#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/unique.h>

#include "../include/exception.cuh"
#include "../include/print.cuh"
#include "../include/relational_algebra.cuh"
#include "../include/timer.cuh"

void RelationalCopy::operator()() {
    GHashRelContainer *src;
    if (src_ver == DELTA) {
        src = src_rel->delta;
    } else {
        src = src_rel->full;
    }
    GHashRelContainer *dest = dest_rel->newt;

    int output_arity = dest_rel->arity;
    column_type *copied_raw_data;
    checkCuda(
        hipMalloc((void **)&copied_raw_data,
                   src->tuple_counts * output_arity * sizeof(column_type)));
    get_copy_result<<<grid_size, block_size>>>(src->tuples, copied_raw_data,
                                               output_arity, src->tuple_counts,
                                               tuple_generator);

    if (dest->tuples == nullptr && dest->tuple_counts == 0) {
        free_relation_container(dest);
        load_relation_container(dest, dest->arity, copied_raw_data,
                                src->tuple_counts, src->index_column_size, 0.8,
                                grid_size, block_size, true, false, false);
    } else {
        GHashRelContainer *tmp =
            new GHashRelContainer(dest->arity, dest->index_column_size);
        load_relation_container(tmp, dest->arity, copied_raw_data,
                                src->tuple_counts, src->index_column_size, 0.8,
                                grid_size, block_size, true, false, false);
        // merge to newt
        GHashRelContainer *old_newt = dest;
        tuple_type *tp_buffer;
        checkCuda(hipMalloc((void **)&tp_buffer,
                             (old_newt->tuple_counts + src->tuple_counts) *
                                 sizeof(tuple_type)));
        tuple_type *tp_buffer_end = thrust::merge(
            thrust::device, old_newt->tuples,
            old_newt->tuples + old_newt->tuple_counts, tmp->tuples,
            tmp->tuples + tmp->tuple_counts, tp_buffer,
            tuple_indexed_less(dest->index_column_size, output_arity));
        checkCuda(hipDeviceSynchronize());
        hipFree(tmp->tuples);
        hipFree(old_newt->tuples);
        tp_buffer_end = thrust::unique(thrust::device, tp_buffer, tp_buffer_end,
                                       t_equal(output_arity));
        checkCuda(hipDeviceSynchronize());
        u64 new_newt_counts = tp_buffer_end - tp_buffer;
        column_type *new_newt_raw;
        checkCuda(hipMalloc((void **)&tp_buffer,
                             (tmp->tuple_counts + old_newt->tuple_counts) *
                                 output_arity * sizeof(column_type)));
        flatten_tuples_raw_data<<<grid_size, block_size>>>(
            tp_buffer, new_newt_raw, new_newt_counts, output_arity);
        checkCuda(hipDeviceSynchronize());
        hipFree(tp_buffer);
        free_relation_container(old_newt);
        free_relation_container(tmp);
        load_relation_container(dest, output_arity, new_newt_raw,
                                new_newt_counts, dest->index_column_size,
                                0.8, grid_size, block_size);
        // delete tmp;
    }
}
